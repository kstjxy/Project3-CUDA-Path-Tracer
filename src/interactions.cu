#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>

// ---------------- Procedural noise helpers (value noise + fBm) ----------------

__host__ __device__ inline float _fract(float x) { return x - floorf(x); }
__host__ __device__ inline float _lerp(float a, float b, float t) { return a + t * (b - a); }
__host__ __device__ inline float _smooth(float t) { return t * t * (3.0f - 2.0f * t); }

__host__ __device__ inline float hash31(const glm::vec3& p)
{
    // Single-value hash from 3D position
    float n = glm::dot(p, glm::vec3(127.1f, 311.7f, 74.7f));
    return _fract(sinf(n) * 43758.5453f);
}

__host__ __device__ float valueNoise3D(const glm::vec3& p)
{
    glm::vec3 i = glm::floor(p);
    glm::vec3 f = p - i;
    glm::vec3 u = glm::vec3(_smooth(f.x), _smooth(f.y), _smooth(f.z));

    // 8 corners
    float n000 = hash31(i + glm::vec3(0, 0, 0));
    float n100 = hash31(i + glm::vec3(1, 0, 0));
    float n010 = hash31(i + glm::vec3(0, 1, 0));
    float n110 = hash31(i + glm::vec3(1, 1, 0));
    float n001 = hash31(i + glm::vec3(0, 0, 1));
    float n101 = hash31(i + glm::vec3(1, 0, 1));
    float n011 = hash31(i + glm::vec3(0, 1, 1));
    float n111 = hash31(i + glm::vec3(1, 1, 1));

    float nx00 = _lerp(n000, n100, u.x);
    float nx10 = _lerp(n010, n110, u.x);
    float nx01 = _lerp(n001, n101, u.x);
    float nx11 = _lerp(n011, n111, u.x);

    float nxy0 = _lerp(nx00, nx10, u.y);
    float nxy1 = _lerp(nx01, nx11, u.y);

    return _lerp(nxy0, nxy1, u.z);
}

__host__ __device__ float fbm(const glm::vec3& p, int octaves)
{
    float v = 0.0f;
    float amp = 0.5f;
    float freq = 1.0f;
    for (int i = 0; i < octaves; ++i)
    {
        v += amp * valueNoise3D(p * freq);
        freq *= 2.0f;
        amp *= 0.5f;
    }
    return v;
}

__host__ __device__ glm::vec3 evalMarble(const glm::vec3& pos, const Material& m)
{
    glm::vec3 pp = pos * m.marbleScale;
    float wobble = m.marbleWarp * fbm(pp, m.marbleOctaves);
    float t = m.marbleFrequency * pp.x + wobble * TWO_PI;
    float s = 0.5f + 0.5f * sinf(t);
    glm::vec3 base = (1.0f - s) * m.marbleColor2 + s * m.marbleColor1;
    return base * m.color; // optional overall tint
}

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ glm::vec3 randomInUnitSphere(
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    while (true)
    {
        float x = u01(rng) * 2.0f - 1.0f;
        float y = u01(rng) * 2.0f - 1.0f;
        float z = u01(rng) * 2.0f - 1.0f;
        glm::vec3 p(x, y, z);
        if (glm::dot(p, p) < 1.0f)
            return p;
    }
}

__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng)
{
    const float bias = EPSILON;
    glm::vec3 wi = glm::normalize(pathSegment.ray.direction);

    // Refractive (glass) material
    if (m.hasRefractive > 0.0f && m.indexOfRefraction > 0.0f)
    {
        // Orientation and indices
        glm::vec3 inDir = wi;
        bool inside = glm::dot(inDir, normal) > 0.0f;
        float etaI = inside ? m.indexOfRefraction : 1.0f;
        float etaT = inside ? 1.0f : m.indexOfRefraction;
        float eta = etaI / etaT;
        glm::vec3 N = inside ? -normal : normal;

        // Directions
        float cosThetaI = glm::clamp(-glm::dot(inDir, N), 0.f, 1.f);
        glm::vec3 reflectDir = glm::reflect(inDir, N);
        glm::vec3 refractDir = glm::refract(inDir, N, eta);

        // Fresnel via Schlick
        float r0 = (etaT - etaI) / (etaT + etaI);
        r0 = r0 * r0;
        float fresnel = r0 + (1.0f - r0) * powf(1.0f - cosThetaI, 5.0f);

        thrust::uniform_real_distribution<float> u01(0, 1);
        bool chooseReflect = (u01(rng) < fresnel) || (glm::dot(refractDir, refractDir) < 1e-12f);

        glm::vec3 newDir = chooseReflect ? reflectDir : refractDir;
        pathSegment.ray.direction = glm::normalize(newDir);
        pathSegment.ray.origin = intersect + (chooseReflect ? N : -N);

        pathSegment.color *= m.color;
        pathSegment.remainingBounces -= 1;
        return;
    }

    // Subsurface scattering
    if (m.hasSubsurface > 0.0f)
    {
        glm::vec3 n = glm::normalize(normal);
        glm::vec3 directionNotNormal = (fabsf(n.x) < SQRT_OF_ONE_THIRD) ? glm::vec3(1, 0, 0)
            : (fabsf(n.y) < SQRT_OF_ONE_THIRD) ? glm::vec3(0, 1, 0) : glm::vec3(0, 0, 1);
        glm::vec3 t1 = glm::normalize(glm::cross(n, directionNotNormal));
        glm::vec3 t2 = glm::normalize(glm::cross(n, t1));

        // mfp = 1 / sigma_t
        glm::vec3 sigmaT = m.sigmaA + m.sigmaS;
        float sigmaT_scalar = fmaxf(1e-6f, (sigmaT.x + sigmaT.y + sigmaT.z) / 3.0f);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float u1 = u01(rng);
        float u2 = u01(rng);
        float rd = 1.0f / sigmaT_scalar;
        float r = -rd * logf(fmaxf(1e-6f, 1.0f - u1));
        float phi = TWO_PI * u2;
        glm::vec3 lateral = r * (cosf(phi) * t1 + sinf(phi) * t2);
        glm::vec3 newPoint = intersect + lateral;

        // Cosine-weighted outgoing direction
        glm::vec3 newDir = glm::normalize(calculateRandomDirectionInHemisphere(n, rng));

        glm::vec3 albedo = glm::vec3(
            sigmaT.x > 0 ? m.sigmaS.x / sigmaT.x : 0.0f,
            sigmaT.y > 0 ? m.sigmaS.y / sigmaT.y : 0.0f,
            sigmaT.z > 0 ? m.sigmaS.z / sigmaT.z : 0.0f);
        glm::vec3 att = glm::exp(-m.sigmaA * r);
        pathSegment.color *= (albedo * att * m.color);

        pathSegment.ray.origin = newPoint + n * bias;
        pathSegment.ray.direction = newDir;
        pathSegment.remainingBounces -= 1;
        return;
    }

    // Reflective / glossy material
    if (m.hasReflective > 0.0f)
    {
        glm::vec3 R = glm::reflect(wi, normal);
        float rough = m.specular.exponent; // interpret exponent as roughness [0..1]
        if (rough > 0.0f)
        {
            glm::vec3 fuzz = rough * randomInUnitSphere(rng);
            R = glm::normalize(R + fuzz);
            if (glm::dot(R, normal) < 0.0f)
            {
                R = glm::reflect(R, normal);
            }
        }
        pathSegment.color *= m.color;
        pathSegment.ray.origin = intersect + normal * bias;
        pathSegment.ray.direction = glm::normalize(R);
        pathSegment.remainingBounces -= 1;
        return;
    }

    // Diffuse scatter (with optional procedural textures)
    glm::vec3 newDir = glm::normalize(calculateRandomDirectionInHemisphere(normal, rng));
    glm::vec3 surfColor = (m.hasMarble > 0.0f) ? evalMarble(intersect, m) : m.color;
    pathSegment.color *= surfColor;
    pathSegment.ray.origin = intersect + normal * bias;
    pathSegment.ray.direction = newDir;
    pathSegment.remainingBounces -= 1;
}
