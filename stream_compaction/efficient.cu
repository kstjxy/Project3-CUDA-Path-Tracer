#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer() { static PerformanceTimer t; return t; }

        __global__ void kernUpSweep(int nPow2, int d, int numOps, int* data) {
            unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i >= (unsigned int)numOps) return;

            unsigned int stride = 1u << (d + 1);
            unsigned int bi = i * stride + (stride - 1u);
            unsigned int ai = bi - (1u << d);
            data[bi] += data[ai];
        }

        __global__ void kernDownSweep(int nPow2, int d, int numOps, int* data) {
            unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i >= (unsigned int)numOps) return;

            unsigned int stride = 1u << (d + 1);
            unsigned int bi = i * stride + (stride - 1u);
            unsigned int ai = bi - (1u << d);

            int t = data[ai];
            data[ai] = data[bi];
            data[bi] += t;
        }

        static void scanInPlace(int* devData, int nPow2) {
            if (nPow2 <= 0) return;

            const int BLOCK_SIZE = 128;
            const int levels = ilog2ceil(nPow2);

            // Up-sweep
            for (int d = 0; d < levels; ++d) {
                int numOps = nPow2 >> (d + 1);
                dim3 block(BLOCK_SIZE);
                dim3 grid((numOps + BLOCK_SIZE - 1) / BLOCK_SIZE);

                kernUpSweep <<<grid, block >>> (nPow2, d, numOps, devData);
                hipDeviceSynchronize();
                checkCUDAError("kernUpSweep");
            }

            hipMemset(devData + (nPow2 - 1), 0, sizeof(int));
            checkCUDAError("hipMemset root");

            // Down-sweep
            for (int d = levels - 1; d >= 0; --d) {
                int numOps = nPow2 >> (d + 1);
                dim3 block(BLOCK_SIZE);
                dim3 grid((numOps + BLOCK_SIZE - 1) / BLOCK_SIZE);

                kernDownSweep <<<grid, block >>> (nPow2, d, numOps, devData);
                hipDeviceSynchronize();
                checkCUDAError("kernDownSweep");
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            if (n <= 0) return;

            const int nPow2 = 1 << ilog2ceil(n);

            int* devData = nullptr;
            hipMalloc(&devData, nPow2 * sizeof(int));
            checkCUDAError("hipMalloc devData");
            hipMemset(devData, 0, nPow2 * sizeof(int));
            checkCUDAError("hipMemset devData");

            hipMemcpy(devData, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("H2D idata");

            timer().startGpuTimer();
            scanInPlace(devData, nPow2);
            timer().endGpuTimer();

            hipMemcpy(odata, devData, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("D2H odata");
            hipFree(devData);
        }

        /**
         * Work-efficient compaction using the same scanInPlace.
         */
        int compact(int n, int* odata, const int* idata) {
            if (n <= 0) return 0;

            const int BLOCK_SIZE = 128;
            dim3 block(BLOCK_SIZE);
            dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            int* devIdata = nullptr, * devBools = nullptr, * devIndices = nullptr, * devOdata = nullptr;

            hipMalloc(&devIdata, n * sizeof(int));
            hipMalloc(&devBools, n * sizeof(int));
            hipMalloc(&devOdata, n * sizeof(int));
            checkCUDAError("hipMalloc inputs");

            hipMemcpy(devIdata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            // Map -> bools
            StreamCompaction::Common::kernMapToBoolean <<<grid, block >>> (n, devBools, devIdata);
            hipDeviceSynchronize();
            checkCUDAError("kernMapToBoolean");

            // Scan bools (exclusive) -> indices
            int nPow2 = 1 << ilog2ceil(n);
            hipMalloc(&devIndices, nPow2 * sizeof(int));
            hipMemset(devIndices, 0, nPow2 * sizeof(int));
            hipMemcpy(devIndices, devBools, n * sizeof(int), hipMemcpyDeviceToDevice);

            scanInPlace(devIndices, nPow2);

            // Scatter
            StreamCompaction::Common::kernScatter <<<grid, block >>> (n, devOdata, devIdata, devBools, devIndices);
            hipDeviceSynchronize();
            checkCUDAError("kernScatter");

            timer().endGpuTimer();

            int lastIdx = 0, lastFlag = 0;
            hipMemcpy(&lastIdx, devIndices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastFlag, devBools + (n - 1), sizeof(int), hipMemcpyDeviceToHost);

            int count = lastIdx + lastFlag;
            if (count > 0) {
                hipMemcpy(odata, devOdata, count * sizeof(int), hipMemcpyDeviceToHost);
            }

            hipFree(devIdata);
            hipFree(devBools);
            hipFree(devIndices);
            hipFree(devOdata);
            return count;
        }
    }
}
